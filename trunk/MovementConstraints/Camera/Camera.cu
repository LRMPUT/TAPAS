/*
 * CameraUtils.cu
 *
 *  Created on: Jul 30, 2014
 *      Author: robots
 */
//#include <opencv2/opencv.hpp>

#include <hip/hip_runtime.h>
//#include <cutil.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>
//#include <hip/hip_vector_types.h>

#include <cstdio>
#include <iostream>

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

#include "CameraKernels.cu"

void cudaAllocateAndCopyToDevice(void** d_dst, void* src, int size){
	checkCudaErrors(hipMalloc(d_dst, size));
	//std::cout << "*d_dst = " << *d_dst << ", src = " << src << ", size = " << size << std::endl;
	checkCudaErrors(hipMemcpy(*d_dst, src, size, hipMemcpyHostToDevice));
}

void cudaCopyFromDeviceAndFree(void* dst, void* d_src, int size){
	checkCudaErrors(hipMemcpy(dst, d_src, size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_src));
}

extern "C" void reprojectCameraPoints(float* invCameraMatrix,
							float* distCoeffs,
							float* curPosCameraMapCenterGlobal,
							float* curPosCameraMapCenterImu,
							int numRows,
							int numCols,
							int* segments,
							int mapSize,
							int rasterSize)
{
	float* d_invCameraMatrix;
	float* d_distCoeffs;
	float* d_curPosCameraMapCenterGlobal;
	float* d_curPosCameraMapCenterImu;
	int* d_segments;

	cudaAllocateAndCopyToDevice((void**)&d_invCameraMatrix,
								invCameraMatrix,
								3*3*sizeof(float));
	//cudaAllocateAndCopyToDevice((void**)&d_distCoeffs,
	//							distCoeffs,
	//							5*sizeof(float));
	cudaAllocateAndCopyToDevice((void**)&d_curPosCameraMapCenterGlobal,
								curPosCameraMapCenterGlobal,
								4*4*sizeof(float));
	cudaAllocateAndCopyToDevice((void**)&d_curPosCameraMapCenterImu,
								curPosCameraMapCenterImu,
								4*4*sizeof(float));
	cudaAllocateAndCopyToDevice((void**)&d_segments,
								segments,
								numRows*numCols*sizeof(int));

	dim3 blockSize(32, 16, 1);
	dim3 gridSize((numCols + blockSize.x - 1) / blockSize.x,
					(numRows + blockSize.y - 1) / blockSize.y);
	compPointReprojection<<<gridSize, blockSize>>>(d_invCameraMatrix,
													d_distCoeffs,
													d_curPosCameraMapCenterGlobal,
													d_curPosCameraMapCenterImu,
													numRows,
													numCols,
													d_segments,
													mapSize,
													rasterSize);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	hipFree(d_invCameraMatrix);
	hipFree(d_distCoeffs);
	hipFree(d_curPosCameraMapCenterGlobal);
	hipFree(d_curPosCameraMapCenterImu);
	cudaCopyFromDeviceAndFree(segments,
								d_segments,
								numRows*numCols*sizeof(int));

}


