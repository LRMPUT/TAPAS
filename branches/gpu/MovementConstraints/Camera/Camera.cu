/*
 * CameraUtils.cu
 *
 *  Created on: Jul 30, 2014
 *      Author: robots
 */
//#include <opencv2/opencv.hpp>

#include <hip/hip_runtime.h>
//#include <cutil.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>
//#include <hip/hip_vector_types.h>

#include <cstdio>
#include <iostream>

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

#include "CameraKernels.cu"
#include "cuPrintf.cu"

void cudaAllocateAndCopyToDevice(void** d_dst, const void* src, int size){
	checkCudaErrors(hipMalloc(d_dst, size));
	//std::cout << "*d_dst = " << *d_dst << ", src = " << src << ", size = " << size << std::endl;
	checkCudaErrors(hipMemcpy(*d_dst, src, size, hipMemcpyHostToDevice));
}

void cudaCopyFromDeviceAndFree(void* dst, void* d_src, int size){
	checkCudaErrors(hipMemcpy(dst, d_src, size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_src));
}

extern "C" void reprojectCameraPoints(float* invCameraMatrix,
							float* distCoeffs,
							float* curPosCameraMapCenterGlobal,
							float* curPosCameraMapCenterImu,
							int numRows,
							int numCols,
							int* segments,
							int mapSize,
							int rasterSize)
{
	float* d_invCameraMatrix;
	float* d_distCoeffs;
	float* d_curPosCameraMapCenterGlobal;
	float* d_curPosCameraMapCenterImu;
	int* d_segments;

	cudaAllocateAndCopyToDevice((void**)&d_invCameraMatrix,
								invCameraMatrix,
								3*3*sizeof(float));
	//cudaAllocateAndCopyToDevice((void**)&d_distCoeffs,
	//							distCoeffs,
	//							5*sizeof(float));
	cudaAllocateAndCopyToDevice((void**)&d_curPosCameraMapCenterGlobal,
								curPosCameraMapCenterGlobal,
								4*4*sizeof(float));
	cudaAllocateAndCopyToDevice((void**)&d_curPosCameraMapCenterImu,
								curPosCameraMapCenterImu,
								4*4*sizeof(float));
	cudaAllocateAndCopyToDevice((void**)&d_segments,
								segments,
								numRows*numCols*sizeof(int));

	dim3 blockSize(32, 16, 1);
	dim3 gridSize((numCols + blockSize.x - 1) / blockSize.x,
					(numRows + blockSize.y - 1) / blockSize.y);
	compPointReprojection<<<gridSize, blockSize>>>(d_invCameraMatrix,
													d_distCoeffs,
													d_curPosCameraMapCenterGlobal,
													d_curPosCameraMapCenterImu,
													numRows,
													numCols,
													d_segments,
													mapSize,
													rasterSize);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipFree(d_invCameraMatrix));
	checkCudaErrors(hipFree(d_distCoeffs));
	checkCudaErrors(hipFree(d_curPosCameraMapCenterGlobal));
	checkCudaErrors(hipFree(d_curPosCameraMapCenterImu));
	cudaCopyFromDeviceAndFree(segments,
								d_segments,
								numRows*numCols*sizeof(int));

}

extern "C" void extractEntries(const unsigned char* const imageH,
								const unsigned char* const imageS,
								const unsigned char* const imageV,
								const float* const terrain,
								const int* const regionsOnImage,
								float* const feat,
								unsigned int* countPixelsEntries,
								unsigned int* countPointsEntries,
								const float* const cameraMatrix,
								const float* const distCoeffs,
								int numRows,
								int numCols,
								int numPoints,
								int numEntries,
								int descLen,
								const FeatParams* const featParams)
{
	unsigned char *d_h, *d_s, *d_v;
	float *d_terrain, *d_feat, *d_cameraMatrix, *d_distCoeffs;
	int *d_segmentsIm, *d_segmentsPoints;
	int* d_featInt;
	unsigned int *d_countSegmentsIm, *d_countSegmentsPoints;
	FeatParams *d_featParams;

	printf("cudaPrintfInit\n");
	cudaPrintfInit(10*1024*1024);
	printf("End cudaPrintfInit\n");

	cudaAllocateAndCopyToDevice((void**)&d_featParams,
									featParams,
									sizeof(FeatParams));
	cudaAllocateAndCopyToDevice((void**)&d_cameraMatrix,
								cameraMatrix,
								3*3*sizeof(float));
	//cudaAllocateAndCopyToDevice((void**)&d_distCoeffs,
	//							distCoeffs,
	//							5*sizeof(float));
	cudaAllocateAndCopyToDevice((void**)&d_h,
								imageH,
								numRows*numCols*sizeof(unsigned char));
	cudaAllocateAndCopyToDevice((void**)&d_s,
									imageS,
									numRows*numCols*sizeof(unsigned char));
	cudaAllocateAndCopyToDevice((void**)&d_v,
									imageV,
									numRows*numCols*sizeof(unsigned char));
	cudaAllocateAndCopyToDevice((void**)&d_terrain,
									(void*)terrain,
									numPoints*sizeof(float));
	cudaAllocateAndCopyToDevice((void**)&d_segmentsIm,
									regionsOnImage,
									numRows*numCols*sizeof(int));
	checkCudaErrors(hipMalloc((void**)&d_segmentsPoints, numPoints*sizeof(int)));

	checkCudaErrors(hipMalloc((void**)&d_feat, numEntries*descLen*sizeof(float)));
	checkCudaErrors(hipMemset(d_feat, 0, numEntries*descLen*sizeof(float)));

	checkCudaErrors(hipMalloc((void**)&d_featInt, numEntries*descLen*sizeof(int)));
	checkCudaErrors(hipMemset(d_featInt, 0, numEntries*descLen*sizeof(int)));

	checkCudaErrors(hipMalloc((void**)&d_countSegmentsIm, numEntries*sizeof(unsigned int)));
	checkCudaErrors(hipMemset(d_countSegmentsIm, 0, numEntries*sizeof(unsigned int)));

	checkCudaErrors(hipMalloc((void**)&d_countSegmentsPoints, numEntries*sizeof(unsigned int)));
	checkCudaErrors(hipMemset(d_countSegmentsPoints, 0, numEntries*sizeof(unsigned int)));

	dim3 blockSizeIm(32, 16, 1);
	dim3 gridSizeIm((numCols + blockSizeIm.x - 1) / blockSizeIm.x,
					(numRows + blockSizeIm.y - 1) / blockSizeIm.y);
	//printf("gridSizeIm = (%d, %d, %d)\n", gridSizeIm.x, gridSizeIm.y, gridSizeIm.z);

	dim3 blockSizePoints(512, 1, 1);
	dim3 gridSizePoints((numPoints + blockSizePoints.x - 1) / blockSizePoints.x, 1, 1);

	dim3 blockSizeEntries(512, 1, 1);
	dim3 gridSizeEntries((numEntries + blockSizeEntries.x - 1) / blockSizeEntries.x, 1, 1);
	//printf("gridSizePoints = (%d, %d, %d)\n", gridSizePoints.x, gridSizePoints.y, gridSizePoints.z);

	//printf("d_segmentsIm = %p, d_countSegmentsIm = %p, numRows = %d, numCols = %d, numEntries = %d\n", d_segmentsIm, d_countSegmentsIm, numRows, numCols, numEntries);
	//printf("d_terrain = %p, d_segmentsPoints = %p\n", d_terrain, d_segmentsPoints);
	//precomputing
	printf("countSegmentPixels\n");
	countSegmentPixels<<<gridSizeIm, blockSizeIm>>>(d_segmentsIm,
													d_countSegmentsIm,
													numRows,
													numCols,
													numEntries);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	if(numPoints > 0){
		printf("compPointProjection\n");
		compPointProjection<<<gridSizePoints, blockSizePoints>>>(d_terrain,
																d_segmentsIm,
																d_segmentsPoints,
																d_cameraMatrix,
																d_distCoeffs,
																numPoints,
																numRows,
																numCols);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());

		printf("countSegmentPoints\n");
		countSegmentPoints<<<gridSizePoints, blockSizePoints>>>(d_segmentsPoints,
																d_countSegmentsPoints,
																numPoints);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
	}

	//extracting features
	int startRow = 0;

	printf("compImageHistHSV\n");
	compImageHistHSV<<<gridSizeIm, blockSizeIm>>>(d_h,
												d_s,
												d_v,
												d_countSegmentsIm,
												d_featInt + startRow*numEntries,
												d_segmentsIm,
												numRows,
												numCols,
												numEntries,
												d_featParams);

	scaleData<<<gridSizeEntries, blockSizeEntries>>>(d_featInt + startRow*numEntries,
													d_feat + startRow*numEntries,
													d_countSegmentsIm,
													featParams->histHLen * featParams->histSLen + featParams->histVLen,
													numEntries);
	cudaPrintfDisplay(stdout, true);
	printf("End displaying cuPrintf\n");
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	startRow += featParams->histHLen * featParams->histSLen + featParams->histVLen;

	printf("compImageMeanHSV\n");
	compImageMeanHSV<<<gridSizeIm, blockSizeIm>>>(d_h,
												d_s,
												d_v,
												d_countSegmentsIm,
												d_featInt + startRow*numEntries,
												d_segmentsIm,
												numRows,
												numCols,
												numEntries,
												d_featParams);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	int meanHSVStartRow = startRow;
	startRow += 3;

	printf("compImageCovarHSV\n");
	compImageCovarHSV<<<gridSizeIm, blockSizeIm>>>(d_h,
													d_s,
													d_v,
													d_countSegmentsIm,
													d_feat + meanHSVStartRow*numEntries,
													d_featInt + startRow*numEntries,
													d_segmentsIm,
													numRows,
													numCols,
													numEntries,
													d_featParams);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	startRow += 9;

	if(numPoints > 0){
		printf("compTerrainHistDI");
		compTerrainHistDI<<<gridSizePoints, blockSizePoints>>>(d_terrain,
																d_segmentsPoints,
																d_countSegmentsPoints,
																d_feat + startRow*numEntries,
																numPoints,
																numEntries,
																d_featParams);
		cudaPrintfDisplay(stdout, true);
		printf("End displaying cuPrintf\n");
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
	}
	startRow += featParams->histDLen * featParams->histILen;

	if(numPoints > 0){
		printf("compTerrainMeanDI\n");
		compTerrainMeanDI<<<gridSizePoints, blockSizePoints>>>(d_terrain,
																d_segmentsPoints,
																d_countSegmentsPoints,
																d_feat + startRow*numEntries,
																numPoints,
																numEntries,
																d_featParams);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
	}
	int meanDIStartRow = startRow;
	startRow += 2;

	if(numPoints > 0){
		printf("compTerrainCovarDI");
		compTerrainCovarDI<<<gridSizePoints, blockSizePoints>>>(d_terrain,
																d_segmentsPoints,
																d_countSegmentsPoints,
																d_feat + meanDIStartRow * numEntries,
																d_feat + startRow*numEntries,
																numPoints,
																numEntries,
																d_featParams);
		hipDeviceSynchronize();
		checkCudaErrors(hipGetLastError());
	}
	startRow += 4;

	cudaPrintfEnd();

	checkCudaErrors(hipFree(d_featParams));
	checkCudaErrors(hipFree(d_cameraMatrix));
	//checkCudaErrors(hipFree(d_distCoeffs));
	checkCudaErrors(hipFree(d_h));
	checkCudaErrors(hipFree(d_s));
	checkCudaErrors(hipFree(d_v));
	checkCudaErrors(hipFree(d_terrain));
	checkCudaErrors(hipFree(d_segmentsIm));
	checkCudaErrors(hipFree(d_segmentsPoints));
	cudaCopyFromDeviceAndFree(feat, d_feat, numEntries*descLen*sizeof(float));
	cudaCopyFromDeviceAndFree(countPixelsEntries, d_countSegmentsIm, numEntries*sizeof(unsigned int));
	cudaCopyFromDeviceAndFree(countPointsEntries, d_countSegmentsPoints, numEntries*sizeof(unsigned int));
}
