#include "hip/hip_runtime.h"

#ifndef CAMERA_KERNELS_CU
#define CAMERA_KERNELS_CU

#include "CameraCuda.h"

__device__ __forceinline__ void multMat(const float* const d_A,
										const float* const d_B,
										float* const d_C,
										int nRowsA,
										int nColsA,
										int nRowsB,
										int nColsB)
{
	int nRowsC = nRowsA;
	int nColsC = nColsB;
	for(int row = 0; row < nRowsA; row++){
		for(int col = 0; col < nColsB; col++){
			d_C[row * nColsC + col] = 0;
			for(int idx = 0; idx < nColsA; idx++){
				d_C[row * nColsC + col] += d_A[row * nColsA + idx]*d_B[idx * nColsB + col];
			}
		}
	}
}


__device__ float atomicAddOwn(float* address, float val)
{
    unsigned int* address_as_uint =
                              (unsigned int*)address;
    unsigned long long int old = *address_as_uint, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_uint, assumed,
                        __float_as_int(val +
                               __int_as_float(assumed)));
    } while (assumed != old);
    return __int_as_float(old);
}


__global__ void compPointReprojection(float* d_invCameraMatrix,
										float* d_distCoeffs,
										float* d_curPosCameraMapCenterGlobal,
										float* d_curPosCameraMapCenterImu,
										int numRows,
										int numCols,
										int* d_segments,
										int mapSize,
										int rasterSize)
{
	//TODO Add distortion coefficients
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;

	if(idxX < numCols && idxY < numRows){
		float pointIm[3] = {idxX,
							idxY,
							1};
		float pointCamNN[3];
		multMat(d_invCameraMatrix, pointIm, pointCamNN, 3, 3, 3, 1);

		float t31 = d_curPosCameraMapCenterGlobal[2*4 + 0];
		float t32 = d_curPosCameraMapCenterGlobal[2*4 + 1];
		float t33 = d_curPosCameraMapCenterGlobal[2*4 + 2];
		float t34 = d_curPosCameraMapCenterGlobal[2*4 + 3];
		float s = (-t34) / (t31 * pointCamNN[0] + t32 * pointCamNN[1] + t33 * pointCamNN[2]); //at z_glob = 0

		float pointCam[4] = {pointCamNN[0]*s,
							pointCamNN[1]*s,
							pointCamNN[2]*s,
							1};
		float pointMapCenter[4];
		multMat(d_curPosCameraMapCenterImu, pointCam, pointMapCenter, 4, 4, 4, 1);

		int xSegm = pointMapCenter[0]/rasterSize + mapSize/2;
		int ySegm = pointMapCenter[1]/rasterSize + mapSize/2;
		//cout << r << ":" << c << " = (" << xSegm << ", " << ySegm << ")" << endl;
		d_segments[idxY * numCols + idxX] = xSegm*mapSize + ySegm;
	}
}


__global__ void countSegmentPixels(const int* const d_segments,
									unsigned int* const d_countSegments,
									int numRows,
									int numCols,
									int numEntries)
{
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx1d = idxY * numCols + idxX;

	if(idxX < numCols && idxY < numRows){
		int segment = d_segments[idx1d];
		if(segment >= 0){
			atomicInc(&d_countSegments[segment], 0xffffffff);
		}
	}
}

__global__ void compPointProjection(const float* const d_terrain,
								const int* const d_imageSeg,
								int* const d_pointSeg,
								const float* const d_cameraMatrix,
								const float* const d_distCoeffs,
								int numPoints,
								int numRows,
								int numCols)
{
	//TODO Add distortion coefficients
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < numPoints){
		float point3d[3] = {d_terrain[0 * numPoints + idx],
							d_terrain[1 * numPoints + idx],
							d_terrain[2 * numPoints + idx]};
		float point2d[3];
		multMat(d_cameraMatrix, point3d, point2d, 3, 3, 3, 1);
		float s = point2d[2];
		point2d[0] /= s;
		point2d[1] /= s;
		point2d[2] /= s;
		int pX = (int)point2d[0];
		int pY = (int)point2d[1];
		if(pX >= 0 && pX < numCols && pY >= 0 && pY < numRows){
			d_pointSeg[idx] = d_imageSeg[pY * numCols + pX];
		}
		else{
			d_pointSeg[idx] = -1;
		}
	}
}

__global__ void countTerrainPixels(const int* const d_segments,
								unsigned int* const d_countSegments,
								int numPoints)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < numPoints){
		int segment = d_segments[idx];
		if(segment >= 0){
			atomicInc(&d_countSegments[segment], 0xffffffff);
		}
	}
}

__global__ void compImageHistHSV(const unsigned char* const d_h,
								const unsigned char* const d_s,
								const unsigned char* const d_v,
								const unsigned int* const d_countSegments,
								float* const d_feat,
								const int* const d_segments,
								int numRows,
								int numCols,
								int numEntries,
								const FeatParams* const d_featParams)
{
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx1d = idxY * numCols + idxX;

	if(idxX < numCols && idxY < numRows){
		int entry = d_segments[idx1d];
		if(entry >= 0){
			int entryCount = d_countSegments[entry];

			int startRow = 0;

			//histHS
			int hBin = (d_h[idx1d] - d_featParams->histHRangeMin) /
					(d_featParams->histHRangeMax - d_featParams->histHRangeMin) * d_featParams->histHLen;
			int sBin = (d_s[idx1d] - d_featParams->histSRangeMin) /
							(d_featParams->histSRangeMax - d_featParams->histSRangeMin) * d_featParams->histSLen;
			atomicAddOwn(&d_feat[(startRow + sBin * d_featParams->histHLen + hBin)* numEntries + entry], 1.0/entryCount);
			startRow += d_featParams->histHLen * d_featParams->histSLen;

			//histV
			int vBin  = (d_v[idx1d] - d_featParams->histVRangeMin) /
					(d_featParams->histVRangeMax - d_featParams->histVRangeMin) * d_featParams->histVLen;
			atomicAddOwn(&d_feat[(startRow + vBin) * numEntries + entry], 1.0/entryCount);
			startRow += d_featParams->histVLen;
		}
	}
}

__global__ void compImageMeanHSV(const unsigned char* const d_h,
								const unsigned char* const d_s,
								const unsigned char* const d_v,
								const unsigned int* const d_countSegments,
								float* const d_feat,
								const int* const d_segments,
								int numRows,
								int numCols,
								int numEntries,
								const FeatParams* const d_featParams)
{
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx1d = idxY * numCols + idxX;

	if(idxX < numCols && idxY < numRows){
		int entry = d_segments[idx1d];
		if(entry >= 0){
			int entryCount = d_countSegments[entry];

			int startRow = 0;
			//meanHSV
			atomicAddOwn(&d_feat[(startRow + 0) * numEntries + entry], (float)d_h[idx1d]/entryCount);
			atomicAddOwn(&d_feat[(startRow + 1) * numEntries + entry], (float)d_s[idx1d]/entryCount);
			atomicAddOwn(&d_feat[(startRow + 2) * numEntries + entry], (float)d_v[idx1d]/entryCount);
			startRow += 3;
		}
	}
}

__global__ void compImageCovarHSV(const unsigned char* const d_h,
								const unsigned char* const d_s,
								const unsigned char* const d_v,
								const unsigned int* const d_countSegments,
								const float* const d_means,
								float* const d_feat,
								const int* const d_segments,
								int numRows,
								int numCols,
								int numEntries,
								const FeatParams* const d_featParams)
{
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx1d = idxY * numCols + idxX;

	if(idxX < numCols && idxY < numRows){
		int entry = d_segments[idx1d];
		if(entry >= 0){
			int entryCount = d_countSegments[entry];

			int startRow = 0;
			//covar HSV
			float c11 = ((float)d_h[idx1d] -  d_means[0 * numEntries + entry])*((float)d_h[idx1d] -  d_means[0 * numEntries + entry])/entryCount;
			float c12 = ((float)d_h[idx1d] -  d_means[0 * numEntries + entry])*((float)d_s[idx1d] -  d_means[1 * numEntries + entry])/entryCount;
			float c13 = ((float)d_h[idx1d] -  d_means[0 * numEntries + entry])*((float)d_v[idx1d] -  d_means[2 * numEntries + entry])/entryCount;
			float c22 = ((float)d_s[idx1d] -  d_means[1 * numEntries + entry])*((float)d_s[idx1d] -  d_means[1 * numEntries + entry])/entryCount;
			float c23 = ((float)d_s[idx1d] -  d_means[1 * numEntries + entry])*((float)d_v[idx1d] -  d_means[2 * numEntries + entry])/entryCount;
			float c33 = ((float)d_v[idx1d] -  d_means[2 * numEntries + entry])*((float)d_v[idx1d] -  d_means[2 * numEntries + entry])/entryCount;
			atomicAddOwn(&d_feat[(startRow + 0) * numEntries + entry], c11);
			atomicAddOwn(&d_feat[(startRow + 1) * numEntries + entry], c12);
			atomicAddOwn(&d_feat[(startRow + 2) * numEntries + entry], c13);
			atomicAddOwn(&d_feat[(startRow + 3) * numEntries + entry], c12);
			atomicAddOwn(&d_feat[(startRow + 4) * numEntries + entry], c22);
			atomicAddOwn(&d_feat[(startRow + 5) * numEntries + entry], c23);
			atomicAddOwn(&d_feat[(startRow + 6) * numEntries + entry], c12);
			atomicAddOwn(&d_feat[(startRow + 7) * numEntries + entry], c23);
			atomicAddOwn(&d_feat[(startRow + 8) * numEntries + entry], c33);
			startRow += 9;
		}
	}
}

__global__ void compTerrainHistDI(const int* const d_terrain,
								const int* const d_segments,
								const unsigned int* const countSegments,
								float* const d_feat,
								int numPoints,
								int numEntries,
								const FeatParams* const d_featParams)
{

}

__global__ void compTerrainMeanDI(const int* const d_terrain,
								const int* const d_segments,
								const unsigned int* const countSegments,
								float* const d_feat,
								int numPoints,
								int numEntries,
								const FeatParams* const d_featParams)
{

}

__global__ void compTerrainCovarDI(const int* const d_terrain,
								const int* const d_segments,
								const unsigned int* const countSegments,
								float* const d_feat,
								int numPoints,
								int numEntries,
								const FeatParams* const d_featParams)
{

}

__global__ void classifySVM(const float* const d_feat,
							int* const predVal,
							int numEntries,
							int numFeat)
{

}

#endif //CAMERA_KERNELS_CU
