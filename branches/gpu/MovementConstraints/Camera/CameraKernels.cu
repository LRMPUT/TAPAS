#include "hip/hip_runtime.h"

#ifndef CAMERA_KERNELS_CU
#define CAMERA_KERNELS_CU

#include "CameraCuda.h"
#include "cuPrintf.cu"

__device__ __forceinline__ void multMat(const float* const d_A,
										const float* const d_B,
										float* const d_C,
										int nRowsA,
										int nColsA,
										int nRowsB,
										int nColsB)
{
	int nRowsC = nRowsA;
	int nColsC = nColsB;
	for(int row = 0; row < nRowsA; row++){
		for(int col = 0; col < nColsB; col++){
			d_C[row * nColsC + col] = 0;
			for(int idx = 0; idx < nColsA; idx++){
				d_C[row * nColsC + col] += d_A[row * nColsA + idx]*d_B[idx * nColsB + col];
			}
		}
	}
}


__device__ float atomicAddFloat(float* address, float val)
{
    unsigned int* address_as_uint =
                              (unsigned int*)address;
    unsigned long long int old = *address_as_uint, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_uint, assumed,
                        __float_as_int(val +
                               __int_as_float(assumed)));
    } while (assumed != old);
    return __int_as_float(old);
}


__global__ void compPointReprojection(float* d_invCameraMatrix,
										float* d_distCoeffs,
										float* d_curPosCameraMapCenterGlobal,
										float* d_curPosCameraMapCenterImu,
										int numRows,
										int numCols,
										int* d_segments,
										int mapSize,
										int rasterSize)
{
	//TODO Add distortion coefficients
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;

	if(idxX < numCols && idxY < numRows){
		float pointIm[3] = {idxX,
							idxY,
							1};
		float pointCamNN[3];
		multMat(d_invCameraMatrix, pointIm, pointCamNN, 3, 3, 3, 1);

		float t31 = d_curPosCameraMapCenterGlobal[2*4 + 0];
		float t32 = d_curPosCameraMapCenterGlobal[2*4 + 1];
		float t33 = d_curPosCameraMapCenterGlobal[2*4 + 2];
		float t34 = d_curPosCameraMapCenterGlobal[2*4 + 3];
		float s = (-t34) / (t31 * pointCamNN[0] + t32 * pointCamNN[1] + t33 * pointCamNN[2]); //at z_glob = 0

		float pointCam[4] = {pointCamNN[0]*s,
							pointCamNN[1]*s,
							pointCamNN[2]*s,
							1};
		float pointMapCenter[4];
		multMat(d_curPosCameraMapCenterImu, pointCam, pointMapCenter, 4, 4, 4, 1);

		int xSegm = pointMapCenter[0]/rasterSize + mapSize/2;
		int ySegm = pointMapCenter[1]/rasterSize + mapSize/2;
		//cout << r << ":" << c << " = (" << xSegm << ", " << ySegm << ")" << endl;
		d_segments[idxY * numCols + idxX] = xSegm*mapSize + ySegm;
	}
}


__global__ void countSegmentPixels(const int* const d_segments,
									unsigned int* const d_countSegments,
									int numRows,
									int numCols,
									int numEntries)
{
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx1d = idxY * numCols + idxX;

	cuPrintf("idx1d = %d\n", idx1d);
	if(idxX < numCols && idxY < numRows){
		int entry = d_segments[idx1d];
		cuPrintf("entry = %d\n", entry);
		if(entry >= 0){
			atomicInc(&d_countSegments[entry], 0xffffffff);
		}
	}
}

__global__ void compPointProjection(const float* const d_terrain,
								const int* const d_imageSeg,
								int* const d_pointSeg,
								const float* const d_cameraMatrix,
								const float* const d_distCoeffs,
								int numPoints,
								int numRows,
								int numCols)
{
	//TODO Add distortion coefficients
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < numPoints){
		float point3d[3] = {d_terrain[0 * numPoints + idx],
							d_terrain[1 * numPoints + idx],
							d_terrain[2 * numPoints + idx]};
		float point2d[3];
		multMat(d_cameraMatrix, point3d, point2d, 3, 3, 3, 1);
		float s = point2d[2];
		point2d[0] /= s;
		point2d[1] /= s;
		point2d[2] /= s;
		int pX = (int)point2d[0];
		int pY = (int)point2d[1];
		if(pX >= 0 && pX < numCols && pY >= 0 && pY < numRows){
			d_pointSeg[idx] = d_imageSeg[pY * numCols + pX];
		}
		else{
			d_pointSeg[idx] = -1;
		}
	}
}

__global__ void countSegmentPoints(const int* const d_segments,
								unsigned int* const d_countSegments,
								int numPoints)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < numPoints){
		int entry = d_segments[idx];
		if(entry >= 0){
			atomicInc(&d_countSegments[entry], 0xffffffff);
		}
	}
}

__global__ void compImageHistHSV(const unsigned char* const d_h,
								const unsigned char* const d_s,
								const unsigned char* const d_v,
								const unsigned int* const d_countSegments,
								float* const d_feat,
								const int* const d_segments,
								int numRows,
								int numCols,
								int numEntries,
								const FeatParams* const d_featParams)
{
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx1d = idxY * numCols + idxX;

	if(idxX < numCols && idxY < numRows){
		int entry = d_segments[idx1d];
		if(entry >= 0){
			int entryCount = d_countSegments[entry];

			int startRow = 0;

			//histHS
			int hBin = (d_h[idx1d] - d_featParams->histHRangeMin) /
					(d_featParams->histHRangeMax - d_featParams->histHRangeMin) * d_featParams->histHLen;
			int sBin = (d_s[idx1d] - d_featParams->histSRangeMin) /
							(d_featParams->histSRangeMax - d_featParams->histSRangeMin) * d_featParams->histSLen;
			atomicAddFloat(&d_feat[(startRow + sBin * d_featParams->histHLen + hBin)* numEntries + entry], 1.0/entryCount);
			startRow += d_featParams->histHLen * d_featParams->histSLen;

			//histV
			int vBin  = (d_v[idx1d] - d_featParams->histVRangeMin) /
					(d_featParams->histVRangeMax - d_featParams->histVRangeMin) * d_featParams->histVLen;
			atomicAddFloat(&d_feat[(startRow + vBin) * numEntries + entry], 1.0/entryCount);
			startRow += d_featParams->histVLen;
		}
	}
}

__global__ void compImageMeanHSV(const unsigned char* const d_h,
								const unsigned char* const d_s,
								const unsigned char* const d_v,
								const unsigned int* const d_countSegments,
								float* const d_feat,
								const int* const d_segments,
								int numRows,
								int numCols,
								int numEntries,
								const FeatParams* const d_featParams)
{
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx1d = idxY * numCols + idxX;

	if(idxX < numCols && idxY < numRows){
		int entry = d_segments[idx1d];
		if(entry >= 0){
			int entryCount = d_countSegments[entry];

			int startRow = 0;
			//meanHSV
			atomicAddFloat(&d_feat[(startRow + 0) * numEntries + entry], (float)d_h[idx1d]/entryCount);
			atomicAddFloat(&d_feat[(startRow + 1) * numEntries + entry], (float)d_s[idx1d]/entryCount);
			atomicAddFloat(&d_feat[(startRow + 2) * numEntries + entry], (float)d_v[idx1d]/entryCount);
			startRow += 3;
		}
	}
}

__global__ void compImageCovarHSV(const unsigned char* const d_h,
								const unsigned char* const d_s,
								const unsigned char* const d_v,
								const unsigned int* const d_countSegments,
								const float* const d_means,
								float* const d_feat,
								const int* const d_segments,
								int numRows,
								int numCols,
								int numEntries,
								const FeatParams* const d_featParams)
{
	int idxX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idxY = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx1d = idxY * numCols + idxX;

	if(idxX < numCols && idxY < numRows){
		int entry = d_segments[idx1d];
		if(entry >= 0){
			int entryCount = d_countSegments[entry];

			int startRow = 0;
			//covar HSV
			float c11 = ((float)d_h[idx1d] -  d_means[0 * numEntries + entry])*((float)d_h[idx1d] -  d_means[0 * numEntries + entry])/entryCount;
			float c12 = ((float)d_h[idx1d] -  d_means[0 * numEntries + entry])*((float)d_s[idx1d] -  d_means[1 * numEntries + entry])/entryCount;
			float c13 = ((float)d_h[idx1d] -  d_means[0 * numEntries + entry])*((float)d_v[idx1d] -  d_means[2 * numEntries + entry])/entryCount;
			float c22 = ((float)d_s[idx1d] -  d_means[1 * numEntries + entry])*((float)d_s[idx1d] -  d_means[1 * numEntries + entry])/entryCount;
			float c23 = ((float)d_s[idx1d] -  d_means[1 * numEntries + entry])*((float)d_v[idx1d] -  d_means[2 * numEntries + entry])/entryCount;
			float c33 = ((float)d_v[idx1d] -  d_means[2 * numEntries + entry])*((float)d_v[idx1d] -  d_means[2 * numEntries + entry])/entryCount;
			atomicAddFloat(&d_feat[(startRow + 0) * numEntries + entry], c11);
			atomicAddFloat(&d_feat[(startRow + 1) * numEntries + entry], c12);
			atomicAddFloat(&d_feat[(startRow + 2) * numEntries + entry], c13);
			atomicAddFloat(&d_feat[(startRow + 3) * numEntries + entry], c12);
			atomicAddFloat(&d_feat[(startRow + 4) * numEntries + entry], c22);
			atomicAddFloat(&d_feat[(startRow + 5) * numEntries + entry], c23);
			atomicAddFloat(&d_feat[(startRow + 6) * numEntries + entry], c12);
			atomicAddFloat(&d_feat[(startRow + 7) * numEntries + entry], c23);
			atomicAddFloat(&d_feat[(startRow + 8) * numEntries + entry], c33);
			startRow += 9;
		}
	}
}

__global__ void compTerrainHistDI(const float* const d_terrain,
								const int* const d_segments,
								const unsigned int* const d_countSegments,
								float* const d_feat,
								int numPoints,
								int numEntries,
								const FeatParams* const d_featParams)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < numPoints){
		int entry = d_segments[idx];
		if(entry >= 0){
			int entryCount = d_countSegments[entry];
			//distance - 4th row
			int dBin = (d_terrain[4 * numPoints + idx] - d_featParams->histDRangeMin) /
								(d_featParams->histDRangeMax - d_featParams->histDRangeMin) * d_featParams->histDLen;
			//intensity - 5th row
			int iBin = (d_terrain[5 * numPoints + idx] - d_featParams->histIRangeMin) /
											(d_featParams->histIRangeMax - d_featParams->histIRangeMin) * d_featParams->histILen;

			int startRow = 0;
			int bin1d = iBin*d_featParams->histDLen + dBin;
			atomicAddFloat(&d_feat[(startRow + bin1d) * numEntries + entry], (float)1.0/entryCount);
			startRow += d_featParams->histDLen * d_featParams->histILen;
		}
	}
}

__global__ void compTerrainMeanDI(const float* const d_terrain,
								const int* const d_segments,
								const unsigned int* const d_countSegments,
								float* const d_feat,
								int numPoints,
								int numEntries,
								const FeatParams* const d_featParams)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < numPoints){
		int entry = d_segments[idx];
		if(entry >= 0){
			int entryCount = d_countSegments[entry];

			int startRow = 0;
			//meanDI
			atomicAddFloat(&d_feat[(startRow + 0) * numEntries + entry], (float)d_terrain[4 * numPoints + idx]/entryCount);
			atomicAddFloat(&d_feat[(startRow + 1) * numEntries + entry], (float)d_terrain[5 * numPoints + idx]/entryCount);
			startRow += 2;
		}
	}
}

__global__ void compTerrainCovarDI(const float* const d_terrain,
								const int* const d_segments,
								const unsigned int* const d_countSegments,
								const float* const d_means,
								float* const d_feat,
								int numPoints,
								int numEntries,
								const FeatParams* const d_featParams)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < numPoints){
		int entry = d_segments[idx];
		if(entry >= 0){
			int entryCount = d_countSegments[entry];

			int startRow = 0;
			//covar DI
			float c11 = (d_terrain[0 * numPoints + idx] - d_means[0 * numEntries + entry]) *
							(d_terrain[0 * numPoints + idx] - d_means[0 * numEntries + entry])/entryCount;
			float c12 = (d_terrain[0 * numPoints + idx] - d_means[0 * numEntries + entry]) *
							(d_terrain[1 * numPoints + idx] - d_means[1 * numEntries + entry])/entryCount;
			float c22 = (d_terrain[1 * numPoints + idx] - d_means[1 * numEntries + entry]) *
							(d_terrain[1 * numPoints + idx] - d_means[1 * numEntries + entry])/entryCount;
			atomicAddFloat(&d_feat[0 * numEntries + entry], c11);
			atomicAddFloat(&d_feat[1 * numEntries + entry], c12);
			atomicAddFloat(&d_feat[2 * numEntries + entry], c12);
			atomicAddFloat(&d_feat[3 * numEntries + entry], c22);
			startRow += 4;
		}
	}
}

__global__ void classifySVM(const float* const d_feat,
							int* const predVal,
							int numEntries,
							int numFeat)
{

}

#endif //CAMERA_KERNELS_CU
